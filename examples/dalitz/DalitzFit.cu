#include "hip/hip_runtime.h"
// ROOT stuff
#include "TRandom.hh"
#include "TCanvas.h" 
#include "TFile.h" 
#include "TH1F.h" 
#include "TH2F.h" 
#include "TStyle.h" 
#include "TRandom3.hh" 
#include "TLegend.h" 
#include "TText.h" 
#include "TLine.h" 

// System stuff
#include <fstream> 
#include <sys/time.h>
#include <sys/times.h>

// GooFit stuff
#include "Variable.hh" 
#include "PolynomialPdf.hh" 
#include "DalitzPlotPdf.hh" 
#include "DalitzVetoPdf.hh" 
#include "ResonancePdf.hh" 
#include "AddPdf.hh"
#include "ProdPdf.hh"
#include "GooPdf.hh" 
#include "FitManager.hh" 
#include "UnbinnedDataSet.hh"

using namespace std;

TCanvas* foo; 
TCanvas* foodal; 
timeval startTime, stopTime, totalTime;
clock_t startCPU, stopCPU; 
tms startProc, stopProc; 
UnbinnedDataSet* data = 0; 

Variable* m12 = 0;
Variable* m13 = 0;
Variable* eventNumber = 0; 
bool fitMasses = false; 
Variable* fixedRhoMass  = new Variable("rho_mass", 0.7758, 0.01, 0.7, 0.8);
Variable* fixedRhoWidth = new Variable("rho_width", 0.1503, 0.01, 0.1, 0.2); 

const fptype _mD0 = 1.86484; 
const fptype _mD02 = _mD0 *_mD0;
const fptype _mD02inv = 1./_mD02; 
const fptype piPlusMass = 0.13957018;
const fptype piZeroMass = 0.1349766;

// Constants used in more than one PDF component. 
Variable* motherM = new Variable("motherM", _mD0);
Variable* chargeM = new Variable("chargeM", piPlusMass);
Variable* neutrlM = new Variable("neutrlM", piZeroMass);
Variable* massSum = new Variable("massSum", _mD0*_mD0 + 2*piPlusMass*piPlusMass + piZeroMass*piZeroMass); // = 3.53481 
Variable* constantOne = new Variable("constantOne", 1); 
Variable* constantZero = new Variable("constantZero", 0); 

GooPdf* kzero_veto = 0; 

fptype cpuGetM23 (fptype massPZ, fptype massPM) {
  return (_mD02 + piZeroMass*piZeroMass + piPlusMass*piPlusMass + piPlusMass*piPlusMass - massPZ - massPM); 
}

void getToyData (std::string toyFileName) {
  TH2F dalitzplot("dalitzplot", "", m12->numbins, m12->lowerlimit, m12->upperlimit, m13->numbins, m13->lowerlimit, m13->upperlimit); 
  std::vector<Variable*> vars;
  vars.push_back(m12);
  vars.push_back(m13);
  vars.push_back(eventNumber); 
  data = new UnbinnedDataSet(vars); 

  int len = 2048;
  char tmp[len];

  std::ifstream reader;
  reader.open(toyFileName.c_str()); 
  std::string buffer;
  while (!reader.eof()) {
    reader >> buffer;
    if (buffer == "====") break; 
    std::cout << buffer; 
  }

  double dummy = 0; 
  while (!reader.eof()) {
    reader.getline (tmp, len, '\n');
    /*
    reader >> dummy;
    reader >> dummy;      // m23, m(pi+ pi-), called m12 in processToyRoot convention. 
    reader >> m12->value; // Already swapped according to D* charge. m12 = m(pi+pi0)
    reader >> m13->value;

    // Errors on Dalitz variables
    reader >> dummy; 
    reader >> dummy; 
    reader >> dummy; 

    reader >> dummy; // Decay time
    reader >> dummy; // sigma_t

    reader >> dummy; // Md0
    reader >> dummy; // deltaM
    reader >> dummy; // ProbSig
    reader >> dummy; // Dst charge
    reader >> dummy; // Run
    reader >> dummy; // Event
    reader >> dummy; // Signal and four bkg fractions. 
    reader >> dummy; 
    reader >> dummy; 
    reader >> dummy; 
    reader >> dummy; 

    // EXERCISE 1 (preliminary): Impose an artificial reconstruction efficiency
    // by throwing out events with a probability linear in m12. 
    // NB! This exercise continues below. 

    // EXERCISE 2: Instead of the above efficiency, impose a 
    // K0 veto, by throwing out events with 0.475 < m23 < 0.505. 

    // EXERCISE 3: Use both the above. 

    //eventNumber->value = data->getNumEvents(); 
    //data->addEvent(); 
    */

    //sscanf the buffer, 20 elements
    //sscanf (tmp, "%lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf", &dummy, &dummy, &m12->value, &m13->value, &dummy, &dummy,
    //	&dummy, &dummy, &dummy, &dummy, &dummy, &dummy, &dummy, &dummy, &dummy, &dummy, &dummy, &dummy, &dummy, &dummy);
    sscanf (tmp, "%lf %lf %lf %lf", &dummy, &dummy, &m12->value, &m13->value);

    std::vector <fptype> list;
    list.push_back (m12->value);
    list.push_back (m13->value);
    list.push_back (data->getNumEvents ());
    data->insertEventVector(list);

    dalitzplot.Fill(m12->value, m13->value); 
  }

#if 0
  dalitzplot.SetStats(false); 
  dalitzplot.Draw("colz");
  foodal->SaveAs("dalitzplot.png"); 
#endif
}

GooPdf* makeKzeroVeto () {
  if (kzero_veto) return kzero_veto; 

  VetoInfo* kVetoInfo = new VetoInfo();
  kVetoInfo->cyclic_index = PAIR_23; 
  kVetoInfo->minimum = new Variable("veto_min", 0.475*0.475);
  kVetoInfo->maximum = new Variable("veto_max", 0.505*0.505);
  vector<VetoInfo*> vetos; vetos.push_back(kVetoInfo); 
  kzero_veto = new DalitzVetoPdf("kzero_veto", m12, m13, motherM, neutrlM, chargeM, chargeM, vetos); 
  return kzero_veto;
}

DalitzPlotPdf* makeSignalPdf (GooPdf* eff = 0) {
  DecayInfo* dtop0pp = new DecayInfo();
  dtop0pp->motherMass  = _mD0; 
  dtop0pp->daug1Mass  = piZeroMass;
  dtop0pp->daug2Mass  = piPlusMass;
  dtop0pp->daug3Mass  = piPlusMass;
  dtop0pp->meson_radius  = 1.5; 
 
  ResonancePdf* rhop  = new ResonancePdf("rhop",
							     new Variable("rhop_amp_real", 1),
							     new Variable("rhop_amp_imag", 0),
							     fixedRhoMass,
							     fixedRhoWidth,
							     1,
							     PAIR_12);


  bool fixAmps = false;

  ResonancePdf* rhom  = new ResonancePdf("rhom", 
							     fixAmps ? new Variable("rhom_amp_real", 0.714) : 
							     new Variable("rhom_amp_real",  0.714, 0.001, 0, 0),
							     fixAmps ? new Variable("rhom_amp_imag", -0.025) :
							     new Variable("rhom_amp_imag", -0.025, 0.1, 0, 0),
							     fixedRhoMass,
							     fixedRhoWidth,
							     1,
							     PAIR_13);

  ResonancePdf* rho0  = new ResonancePdf("rho0", 
							     fixAmps ? new Variable("rho0_amp_real", 0.565) : 
							     new Variable("rho0_amp_real", 0.565, 0.001, 0, 0),
							     fixAmps ? new Variable("rho0_amp_imag", 0.164) :
							     new Variable("rho0_amp_imag", 0.164, 0.1, 0, 0),
							     fixedRhoMass,
							     fixedRhoWidth,
							     1,
							     PAIR_23);

  Variable* sharedMass = new Variable("rhop_1450_mass", 1.465, 0.01, 1.0, 2.0);
  Variable* shareWidth = new Variable("rhop_1450_width", 0.400, 0.01, 0.01, 5.0); 

  ResonancePdf* rhop_1450  = new ResonancePdf("rhop_1450", 
								  fixAmps ? new Variable("rhop_1450_amp_real", -0.174) : 
								  new Variable("rhop_1450_amp_real", -0.174, 0.001, 0, 0),
								  fixAmps ? new Variable("rhop_1450_amp_imag", -0.117) :
								  new Variable("rhop_1450_amp_imag", -0.117, 0.1, 0, 0),
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_12);

  ResonancePdf* rho0_1450  = new ResonancePdf("rho0_1450", 
								  fixAmps ? new Variable("rho0_1450_amp_real", 0.325) : 
								  new Variable("rho0_1450_amp_real", 0.325, 0.001, 0, 0),
								  fixAmps ? new Variable("rho0_1450_amp_imag", 0.057) : 
								  new Variable("rho0_1450_amp_imag", 0.057, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_23);

  ResonancePdf* rhom_1450  = new ResonancePdf("rhom_1450", 
								  fixAmps ? new Variable("rhom_1450_amp_real", 0.788) : 
								  new Variable("rhom_1450_amp_real", 0.788, 0.001, 0, 0),
								  fixAmps ? new Variable("rhom_1450_amp_imag", 0.226) : 
								  new Variable("rhom_1450_amp_imag", 0.226, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_13);

  sharedMass = new Variable("rhop_1700_mass",  1.720, 0.01, 1.6, 1.9);
  shareWidth = new Variable("rhop_1700_width", 0.250, 0.01, 0.1, 1.0); 

  
  ResonancePdf* rhop_1700  = new ResonancePdf("rhop_1700", 
								  fixAmps ? new Variable("rhop_1700_amp_real", 2.151) : 
								  new Variable("rhop_1700_amp_real",  2.151, 0.001, 0, 0),
								  fixAmps ? new Variable("rhop_1700_amp_imag", -0.658) : 
								  new Variable("rhop_1700_amp_imag", -0.658, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_12);
  
  ResonancePdf* rho0_1700  = new ResonancePdf("rho0_1700", 
								  fixAmps ? new Variable("rho0_1700_amp_real",  2.400) : 
								  new Variable("rho0_1700_amp_real",  2.400, 0.001, 0, 0),
								  fixAmps ? new Variable("rho0_1700_amp_imag", -0.734) : 
								  new Variable("rho0_1700_amp_imag", -0.734, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_23);
  
  ResonancePdf* rhom_1700  = new ResonancePdf("rhom_1700", 
								  fixAmps ? new Variable("rhom_1700_amp_real",  1.286) : 
								  new Variable("rhom_1700_amp_real",  1.286, 0.001, 0, 0),
								  fixAmps ? new Variable("rhom_1700_amp_imag", -1.532) : 
								  new Variable("rhom_1700_amp_imag", -1.532, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_13);
  
  ResonancePdf* f0_980  = new ResonancePdf("f0_980", 
							       fixAmps ? new Variable("f0_980_amp_real",  0.008 * (-_mD02)) : 
							       new Variable("f0_980_amp_real",  0.008 * (-_mD02), 0.001, 0, 0),
							       fixAmps ? new Variable("f0_980_amp_imag", -0.013 * (-_mD02)) : 
							       new Variable("f0_980_amp_imag", -0.013 * (-_mD02), 0.1, 0, 0),  
							       new Variable("f0_980_mass",     0.980, 0.01, 0.8, 1.2),
							       new Variable("f0_980_width",    0.044, 0.001, 0.001, 0.08),
							       0,
							       PAIR_23);
  
  ResonancePdf* f0_1370  = new ResonancePdf("f0_1370", 
								fixAmps ? new Variable("f0_1370_amp_real", -0.058 * (-_mD02)) : 
								new Variable("f0_1370_amp_real", -0.058 * (-_mD02), 0.001, 0, 0),
								fixAmps ? new Variable("f0_1370_amp_imag",  0.026 * (-_mD02)) : 
								new Variable("f0_1370_amp_imag",  0.026 * (-_mD02), 0.1, 0, 0),  
								new Variable("f0_1370_mass",     1.434, 0.01, 1.2, 1.6),
								new Variable("f0_1370_width",    0.173, 0.01, 0.01, 0.4),
								0,
								PAIR_23);
  
  ResonancePdf* f0_1500  = new ResonancePdf("f0_1500", 
								fixAmps ? new Variable("f0_1500_amp_real", 0.057 * (-_mD02)) : 
								new Variable("f0_1500_amp_real", 0.057 * (-_mD02), 0.001, 0, 0),
								fixAmps ? new Variable("f0_1500_amp_imag", 0.012 * (-_mD02)) : 
								new Variable("f0_1500_amp_imag", 0.012 * (-_mD02), 0.1, 0, 0),  
								new Variable("f0_1500_mass",     1.507, 0.01, 1.3, 1.7),
								new Variable("f0_1500_width",    0.109, 0.01, 0.01, 0.3),
								0,
								PAIR_23);
  
  ResonancePdf* f0_1710  = new ResonancePdf("f0_1710", 
								fixAmps ? new Variable("f0_1710_amp_real", 0.070 * (-_mD02)) : 
								new Variable("f0_1710_amp_real", 0.070 * (-_mD02), 0.001, 0, 0),
								fixAmps ? new Variable("f0_1710_amp_imag", 0.087 * (-_mD02)) : 
								new Variable("f0_1710_amp_imag", 0.087 * (-_mD02), 0.1, 0, 0),  
								new Variable("f0_1710_mass",     1.714, 0.01, 1.5, 2.9), 
								new Variable("f0_1710_width",    0.140, 0.01, 0.01, 0.5),
								0,
								PAIR_23);
  
  ResonancePdf* f2_1270  = new ResonancePdf("f2_1270", 
								fixAmps ? new Variable("f2_1270_amp_real", -1.027 * (-_mD02inv)) : 
								new Variable("f2_1270_amp_real", -1.027 * (-_mD02inv), 0.001, 0, 0),
								fixAmps ? new Variable("f2_1270_amp_imag", -0.162 * (-_mD02inv)) : 
								new Variable("f2_1270_amp_imag", -0.162 * (-_mD02inv), 0.1, 0, 0),  
								new Variable("f2_1270_mass",     1.2754, 0.01, 1.0, 1.5),
								new Variable("f2_1270_width",    0.1851, 0.01, 0.01, 0.4),
								2,
								PAIR_23);
  
  ResonancePdf* f0_600  = new ResonancePdf("f0_600", 
							       fixAmps ? new Variable("f0_600_amp_real", 0.068 * (-_mD02)) : 
							       new Variable("f0_600_amp_real", 0.068 * (-_mD02), 0.001, 0, 0),
							       fixAmps ? new Variable("f0_600_amp_imag", 0.010 * (-_mD02)) : 
							       new Variable("f0_600_amp_imag", 0.010 * (-_mD02), 0.1, 0, 0),  
							       new Variable("f0_600_mass",     0.500, 0.01, 0.3, 0.7),
							       new Variable("f0_600_width",    0.400, 0.01, 0.2, 0.6), 
							       0,
							       PAIR_23);
  
  ResonancePdf* nonr  = new ResonancePdf("nonr",
							     fixAmps ? new Variable("nonr_amp_real", 0.5595 * (-1)) : 
							     new Variable("nonr_amp_real", 0.5595 * (-1),   0.001, 0, 0),
							     fixAmps ? new Variable("nonr_amp_imag", -0.108761 * (-1)) : 
							     new Variable("nonr_amp_imag", -0.108761* (-1), 0.1, 0, 0)); 

  dtop0pp->resonances.push_back(nonr); 
  dtop0pp->resonances.push_back(rhop);
  dtop0pp->resonances.push_back(rho0); 
  dtop0pp->resonances.push_back(rhom); 
  dtop0pp->resonances.push_back(rhop_1450); 
  dtop0pp->resonances.push_back(rho0_1450); 
  dtop0pp->resonances.push_back(rhom_1450); 
  dtop0pp->resonances.push_back(rhop_1700); 
  dtop0pp->resonances.push_back(rho0_1700); 
  dtop0pp->resonances.push_back(rhom_1700); 
  dtop0pp->resonances.push_back(f0_980); 
  dtop0pp->resonances.push_back(f0_1370); 
  dtop0pp->resonances.push_back(f0_1500); 
  dtop0pp->resonances.push_back(f0_1710); 
  dtop0pp->resonances.push_back(f2_1270); 
  dtop0pp->resonances.push_back(f0_600); 

  if (!fitMasses) {
    for (vector<ResonancePdf*>::iterator res = dtop0pp->resonances.begin(); res != dtop0pp->resonances.end(); ++res) {
      (*res)->setParameterConstantness(true); 
    }
  }

  if (!eff) {
    // By default create a constant efficiency. 
    vector<Variable*> offsets;
    vector<Variable*> observables;
    vector<Variable*> coefficients; 

    observables.push_back(m12);
    observables.push_back(m13);
    offsets.push_back(constantZero);
    offsets.push_back(constantZero);
    coefficients.push_back(constantOne); 
    eff = new PolynomialPdf("constantEff", observables, coefficients, offsets, 0);
  }

  return new DalitzPlotPdf("signalPDF", m12, m13, eventNumber, dtop0pp, eff);
}

void runToyFit (std::string toyFileName) {
  m12 = new Variable("m12", 0, 3);
  m13 = new Variable("m13", 0, 3); 
  m12->numbins = 240;
  m13->numbins = 240;
  eventNumber = new CountVariable("eventNumber", 0, INT_MAX);
  getToyData(toyFileName);

  // EXERCISE 1 (real part): Create a PolynomialPdf which models
  // the efficiency you imposed in the preliminary, and use it in constructing
  // the signal PDF. 

  // EXERCISE 2: Create a K0 veto function and use it as the efficiency. 

  // EXERCISE 3: Make the efficiency a product of the two functions
  // from the previous exercises.

  DalitzPlotPdf* signal = makeSignalPdf(); 
  signal->setData(data); 
  signal->setDataSize(data->getNumEvents()); 
  FitManager datapdf(signal); 
  
  gettimeofday(&startTime, NULL);
  startCPU = times(&startProc);

  datapdf.setMaxCalls (10);
  datapdf.fit();
  datapdf.getMinuitValues();
  std::vector<Variable*> modParams;
  signal->getParameters(modParams);

  std::vector<double> expected;  //GooFit values
  expected.push_back(-5.59500e-01);
  expected.push_back(1.08761e-01);
  expected.push_back(1);
  expected.push_back(0);
  expected.push_back(5.65000e-01);
  expected.push_back(1.64000e-01);
  expected.push_back(7.14000e-01);
  expected.push_back(-2.50000e-02);;
  expected.push_back(-1.74000e-01);
  expected.push_back(-1.17000e-01);
  expected.push_back(3.25000e-01);
  expected.push_back(5.70000e-02);
  expected.push_back(7.88000e-01);
  expected.push_back(2.26000e-01);
  expected.push_back(2.15100);
  expected.push_back(-6.58000e-01);
  expected.push_back(2.40000);
  expected.push_back(-7.34000e-01);
  expected.push_back(1.28600);
  expected.push_back(-1.53200);
  expected.push_back(-2.78210e-02);
  expected.push_back(4.52092e-02);
  expected.push_back(2.01702e-01);
  expected.push_back(-9.04183e-02);
  expected.push_back(-1.98225e-01);
  expected.push_back(-4.17315e-02);
  expected.push_back(-2.43434e-01);
  expected.push_back(-3.02554e-01);
  expected.push_back(2.95316e-01);
  expected.push_back(4.65835e-02);
  expected.push_back(-2.36479e-01);
  expected.push_back(-3.47763e-02);
  expected.push_back(7.75800e-01);
  expected.push_back(1.50300e-01);
  expected.push_back(1.46500);
  expected.push_back(4.00000e-01);
  expected.push_back(1.72000);
  expected.push_back(2.50000e-01);
  expected.push_back(9.80000e-01);
  expected.push_back(4.40000e-02);
  expected.push_back(1.43400);
  expected.push_back(1.73000e-01);
  expected.push_back(1.50700);
  expected.push_back(1.09000e-01);
  expected.push_back(1.71400);
  expected.push_back(1.40000e-01);
  expected.push_back(1.27540);
  expected.push_back(1.85100e-01);
  expected.push_back(5.00000e-01);
  expected.push_back(4.00000e-01);
  expected.push_back(0);
  expected.push_back(1);  
  
  double variation;
  int count = 0;
  for (int i = 0; i < modParams.size(); i++) { 
    variation = fabs(expected[i] - modParams[i]->value); //expected - actual
    //check the variance of the generated parameter from its actual value and compare it to our epsilon of 0.001
    if (variation > 0.001) {
      std::cout << "\n" << modParams[i]->name << " value not in epsilon." << endl;
      std::cout << "Expected: " << expected[i] << endl;
      std::cout << "Actual: " << modParams[i]->value << endl;
      std::cout << "Variation: " << variation << endl;
      count++;
    }
  }
  
  std::cout << "\nTotal variances: " << count << endl;  
  stopCPU = times(&stopProc);
  gettimeofday(&stopTime, NULL);
}

int main (int argc, char** argv) {
#ifdef TARGET_MPI
  MPI_Init(&argc, &argv);

  //we have MPI, so lets do something slightly different here:
  int myId, numProcs;
  MPI_Comm_size(MPI_COMM_WORLD, &numProcs);
  MPI_Comm_rank(MPI_COMM_WORLD, &myId);

#ifndef TARGET_OMP
  //set the processes to gpus here
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  
  //No way to figure out how many processes per node, so we read the environment variable
  int nodes = atoi (getenv ("PBS_NUM_NODES"));
  if (nodes == 0)
    nodes = 1;
  int procsPerNode = numProcs/nodes;
  int localRank = myId % procsPerNode;

  if (deviceCount == 1 && localRank > 1)
  {
    printf ("Multi-process to one GPU!\n");
    hipSetDevice (0);
  }
  else if (procsPerNode > 1 && deviceCount > 1)
  {
     if (localRank <= deviceCount)
     {
       printf ("setting multiple processes to multiple GPU's\n");
       hipSetDevice (localRank);
     }
     else
     {
       printf ("More multi-processes than multi-gpu's!\n");
       hipSetDevice (localRank % deviceCount);
     }
  }
  else
  {
    printf ("Multi-GPU's, using one process! %i, [%i,%i]\n", deviceCount, localRank, procsPerNode);
    hipSetDevice (0);
  }
#endif
#endif

 //check to see that the file exists
 ifstream ifile(argv[1]);
 if (ifile) {

  gStyle->SetCanvasBorderMode(0);
  gStyle->SetCanvasColor(10);
  gStyle->SetFrameFillColor(10);
  gStyle->SetFrameBorderMode(0);
  gStyle->SetPadColor(0);
  gStyle->SetTitleColor(1);
  gStyle->SetStatColor(0);
  gStyle->SetFillColor(0);
  gStyle->SetFuncWidth(1);
  gStyle->SetLineWidth(1);
  gStyle->SetLineColor(1);
  gStyle->SetPalette(1, 0);
  foo = new TCanvas(); 
  foodal = new TCanvas(); 
  foodal->Size(10, 10);

  runToyFit(argv[1]);

  // Print total minimization time
  double myCPU = stopCPU - startCPU;
  double totalCPU = myCPU; 

  timersub(&stopTime, &startTime, &totalTime);
  std::cout << "Wallclock time  : " << totalTime.tv_sec + totalTime.tv_usec/1000000.0 << " seconds." << std::endl;
  std::cout << "CPU time: " << (myCPU / CLOCKS_PER_SEC) << std::endl; 
  std::cout << "Total CPU time: " << (totalCPU / CLOCKS_PER_SEC) << std::endl; 
  myCPU = stopProc.tms_utime - startProc.tms_utime;
  std::cout << "Processor time: " << (myCPU / CLOCKS_PER_SEC) << std::endl;

#ifdef TARGET_MPI
  MPI_Finalize();
#endif

  delete m12;
  delete m13;
  delete eventNumber;
  delete foo;
  delete foodal;
  delete constantOne;
  delete constantZero;
  delete fixedRhoMass;
  delete fixedRhoWidth;
  delete motherM;
  delete chargeM;
  delete neutrlM;
  delete massSum;
  delete data;
 
 } else {
    std::cout << "**ERROR: File not found**" << endl;
 }
  return 0; 
}
