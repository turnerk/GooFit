#include "hip/hip_runtime.h"
#include "Variable.hh" 
#include "../../PDFs/GaussianPdf.hh" 
#include "../../FitManager.hh" 
#include "../../UnbinnedDataSet.hh" 
#include "../../PDFs/GooPdf.hh"
#include "../../PdfBase.hh"

#include "TRandom.hh" 
#include "TH1F.h"
#include "TCanvas.h" 

#include <sys/time.h>
#include <sys/times.h>
#include <stdio.h>	

using namespace std; 

int main (int argc, char** argv) {
  
#ifdef TARGET_MPI
  MPI_Init(&argc, &argv);

  //we have MPI, so lets do something slightly different here:
  int myId, numProcs;
  MPI_Comm_size(MPI_COMM_WORLD, &numProcs);
  MPI_Comm_rank(MPI_COMM_WORLD, &myId);

#ifndef TARGET_OMP
  //set the processes to gpus here
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  
  //No way to figure out how many processes per node, so we read the environment variable
  int nodes = atoi (getenv ("PBS_NUM_NODES"));
  if (nodes == 0)
    nodes = 1;
  int procsPerNode = numProcs/nodes;
  int localRank = myId % procsPerNode;

  if (deviceCount == 1 && localRank > 1)
  {
    printf ("Multi-process to one GPU!\n");
    hipSetDevice (0);
  }
  else if (procsPerNode > 1 && deviceCount > 1)
  {
     if (localRank <= deviceCount)
     {
       printf ("setting multiple processes to multiple GPU's\n");
       hipSetDevice (localRank);
     }
     else
     {
       printf ("More multi-processes than multi-gpu's!\n");
       hipSetDevice (localRank % deviceCount);
     }
  }
  else
  {
    printf ("Multi-GPU's, using one process! %i, [%i,%i]\n", deviceCount, localRank, procsPerNode);
    hipSetDevice (0);
  }
#endif
#endif


  Variable* xvar = new Variable("xvar", -5, 5); 

  int numbins = 10000;

  // Generate data
  TRandom donram(42); 
  UnbinnedDataSet data(xvar); // Stores events
  for (int i = 0; i < numbins; ++i) {
    fptype val = donram.Gaus(0.2, 1.1); // these are the values for mean and sigma
    if (fabs(val) > 5) {
	--i; 
	continue;
    } 
    data.addEvent(val); 
  }

  // Create the PDF
  Variable* mean = new Variable("mean", 0, 1, -10, 10);
  Variable* sigma = new Variable("sigma", 1, 0.5, 1.5); 
  GaussianPdf* gauss = new GaussianPdf("gauss", xvar, mean, sigma); 

  timeval startTime, stopTime, totalTime;

  // Run a fit
  gauss->setData(&data);
  FitManager datapdf(gauss); 
  gettimeofday(&startTime, NULL);
  datapdf.fit(); 
  
  // Redirect the output to 'output.txt'
  freopen("output.txt","w",stdout);

  fptype* host_output = new fptype[xvar->numbins];
  gauss->transformGrid(host_output);
  std::cout << "TRANSFORMGRID: \n" << std::endl;
  for (int i = 0; i < xvar->numbins; i++) {
    std::cout << host_output[i] << std::endl;
  }
    
  vector<fptype> res;
  gauss->evaluateAtPoints(xvar, res);
  std::cout << "\nEVALUATEATPOINTS: \n" << std::endl;
  for (int i = 0; i < res.size(); i++) {
      std::cout <<  res[i] << std::endl;
  }
     
  vector<vector<fptype> > values;
  gauss->getCompProbsAtDataPoints(values);
  std::cout << "\nGETCOMPPROBSATDATAPOINTS: \n" << std::endl;
  for (int i = 0; i < values.size(); i++) {
    for (int j = 0; j < values.size(); j++) {
      std::cout << values[i][j] << std::endl;
    }
  }
  
  gettimeofday(&stopTime, NULL);

  timersub(&stopTime, &startTime, &totalTime);
  std::cout << "\nWallclock time  : " << totalTime.tv_sec + totalTime.tv_usec/1000000.0 << " seconds." << std::endl;

  return 0;
}
