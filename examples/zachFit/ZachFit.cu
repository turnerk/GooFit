#include "hip/hip_runtime.h"
// ROOT stuff
#include "TRandom.h"
#include "TCanvas.h" 
#include "TFile.h" 
#include "TH1F.h" 
#include "TStyle.h" 

// System stuff
#include <fstream> 
#include <sys/time.h>
#include <sys/times.h>

// RooFit stuff
#include "RooRealVar.h" 
#include "RooDataSet.h"
#include "RooArgSet.h"
#include "RooPlot.h" 
#include "RooGaussian.h"
#include "RooAddPdf.h"
#include "RooMinuit.h" 
#include "RooNLLVar.h" 

// GooFit stuff
#include "Variable.hh" 
#include "KinLimitBWPdf.hh" 
#include "ConvolutionPdf.hh"
#include "GaussianPdf.hh"
#include "ScaledGaussianPdf.hh"
#include "ArgusPdf.hh"
#include "AddPdf.hh"
#include "FitManager.hh" 

#ifdef CUDAPRINT
#include "cuPrintf.cuh" 
#endif 

TCanvas* foo; 
timeval startTime, stopTime, totalTime;
clock_t startCPU, stopCPU; 
tms startProc, stopProc; 
BinnedDataSet* binnedData = 0; 
UnbinnedDataSet* data = 0; 
int length = 0;

TH1F* data_hist = 0;
Variable* dm;

double pdf_int;

char histName[1000];
int numHists = 0; 
#ifdef OMP_ON
#pragma omp threadprivate (numHists, histName)
#endif

TH1F* plotComponent (GooPdf* toPlot, double normFactor) {
//  static char name[1000];
//  static int numHists = 0; 
#ifdef OMP_ON
  sprintf(histName, "%s_hist_%i_%i", toPlot->getName().c_str(), numHists++, omp_get_thread_num());
#else
  sprintf(histName, "%s_hist_%i", toPlot->getName().c_str(), numHists++);
#endif
  TH1F* ret = new TH1F(histName, "", dm->numbins, dm->lowerlimit, dm->upperlimit);
  std::vector<fptype> binValues;
  toPlot->evaluateAtPoints(dm, binValues); 

  pdf_int = 0;
  double step = dm->upperlimit - dm->lowerlimit;
  step /= dm->numbins; 
  for (int i = 1; i <= dm->numbins; ++i) {
    //std::cout << name << " " << i << " : " << binValues[i-1] << " " << (dm->lowerlimit + (i-1)*step) << std::endl;
    pdf_int += binValues[i-1];
  } 
  for (int i = 1; i <= dm->numbins; ++i) ret->SetBinContent(i, binValues[i-1] * normFactor / pdf_int);
  return ret; 
}

void getMCData () {
  data = new UnbinnedDataSet(dm); 
  ifstream mcreader;
  mcreader.open("../../dataFiles/dstwidth_kpi_resMC.dat"); // open the stream
  if (!mcreader.good()) {
    cout << "Error reading from file." << endl;
    exit(-1);
  }

  TH1F* mchist = new TH1F("mchist", "", 300, 0.1365, 0.1665);

  double currDM = 0; 
  while (true) {
    mcreader >> currDM;
    if (mcreader.eof()) break; 
    if (currDM < 0.13957) std::cout << "Bad DM\n"; 
    data->addEvent(currDM);
    mchist->Fill(currDM); 
  }

  mchist->SetStats(false); 
  mchist->SetMarkerStyle(8);
  mchist->SetMarkerSize(0.6); 
  mchist->Draw("p"); 

  foo->SetLogy(true); 
  foo->SaveAs("zach_mchist.png"); 

  std::cout << "MC: Got " << data->getNumEvents() << " events.\n"; 
  mcreader.close(); // close the stream
}

void getData () {
  ifstream datareader;
  datareader.open("../../dataFiles/dstwidth_kpi_data.dat"); // open the stream
  if (!datareader.good()) {
    cout << "Error reading from file." << endl;
    exit(-1);
  }

  binnedData = new BinnedDataSet(dm); 
  delete data;
  data = new UnbinnedDataSet(dm); 
  double currDM = 0; 
  while (true) {
    datareader >> currDM;
    if (datareader.eof()) break; 
    if (currDM > dm->upperlimit) continue;
    if (currDM < dm->lowerlimit) continue;
    data->addEvent(currDM);
    data_hist->Fill(currDM); 

    binnedData->addEvent(currDM); 
  }

  std::cout << "Data events: " << data->getNumEvents() << std::endl; 
  datareader.close(); // close the stream
}

void CudaMinimise (int dev, int fitType) {
#ifdef CUDAPRINT
  cudaPrintfInit(10000000); 
#endif
//#ifdef 0 
#if 0
  int deviceCount;  
  int threadCount;  
#pragma omp parallel
  {
  threadCount = omp_get_num_threads();
  }

  hipGetDeviceCount(&deviceCount);   
  if (threadCount > deviceCount) {
    omp_set_num_threads(deviceCount); 
  }

#pragma omp parallel
  {
    int tid;
    hipDeviceProp_t deviceProp;
    tid = omp_get_thread_num();
    hipGetDeviceProperties(&deviceProp, tid);
    printf("Device %d has compute capability %d.%d.\n",
       tid, deviceProp.major, deviceProp.minor);
    if (deviceProp.major < 2) {
       printf("Compute capability of device %d is less than 2.0, terminating ...\n");
       exit(EXIT_FAILURE);
    }
    hipSetDevice(tid);
#else
  printf ("set device\n"); 
  //hipSetDevice(dev);
#endif

//#ifdef 0
#if 0
#pragma omp master
{
#endif
  dm = new Variable("dm", 0.1395, 0.1665); 
  dm->numbins = 2700; 
  //dm->numbins = 540; 

  printf ("getMCData\n"); 
  getMCData();
  std::cout << "Done getting MC\n"; 
//#ifdef 0
#if 0
}
#pragma omp barrier
#endif

  //we have MPI, so lets do something slightly different here:
  //int deviceCount;
  //hipGetDeviceCount(&deviceCount);

#ifdef TARGET_MPI
  int myId, numProcs;
  MPI_Comm_size(MPI_COMM_WORLD, &numProcs);
  MPI_Comm_rank(MPI_COMM_WORLD, &myId);

  //No way to figure out how many processes per node, so we read the environment variable
  int nodes = atoi (getenv ("PBS_NUM_NODES"));
  if (nodes == 0)
    nodes = 1;
  int procsPerNode = numProcs/nodes;
  int localRank = myId % procsPerNode;

  /*
  if (deviceCount == 1 && localRank > 1)
  {
    printf ("Multi-process to one GPU!\n");
    hipSetDevice (0);
  }
  else if (procsPerNode > 1 && deviceCount > 1)
  {
     if (localRank <= deviceCount)
     {
       printf ("setting multiple processes to multiple GPU's\n");
       hipSetDevice (localRank);
     }
     else
     {
       printf ("More multi-processes than multi-gpu's!\n");
       hipSetDevice (localRank % deviceCount);
     }
  }
  else
  {
    printf ("Multi-GPU's, using one process! %i, [%i,%i]\n", deviceCount, localRank, procsPerNode);
    hipSetDevice (0);
  }
  */ 
#endif

  Variable mean1("kpi_mc_mean1", 0.145402, 0.00001, 0.143, 0.148);
  Variable mean2("kpi_mc_mean2", 0.145465, 0.00001, 0.145, 0.1465);
  Variable mean3("kpi_mc_mean3", 0.145404, 0.00001, 0.144, 0.147);

  Variable sigma1("kpi_mc_sigma1", 0.00010, 0.00001, 0.000001, 0.002);
  Variable sigma2("kpi_mc_sigma2", 0.00075, 0.00001, 0.000001, 0.005);
  Variable sigma3("kpi_mc_sigma3", 0.00020, 0.00001, 0.000005, 0.001);

  Variable pimass("kpi_mc_pimass", 0.13957);
  Variable aslope("kpi_mc_aslope", -20.0, 1, -100.0, 10.0);
  Variable apower("kpi_mc_apower", 1.3, 0.1, 0.1, 10.0);
  Variable gfrac1("kpi_mc_gfrac1", 0.65, 0.01, 0.0, 0.9);
  Variable gfrac2("kpi_mc_gfrac2", 0.02, 0.001, 0.0, 0.12);
  Variable afrac("kpi_mc_afrac", 0.005, 0.003, 0.0, 0.1);

  //mean1.fixed = true;
  //mean2.fixed = true;
  //mean3.fixed = true;
  //sigma1.fixed = true;
  //sigma2.fixed = true;
  //sigma3.fixed = true;

  //aslope.fixed = true;
  //apower.fixed = true;
  //gfrac1.fixed = true;
  //gfrac2.fixed = true;
  //afrac.fixed = true; 

  GaussianPdf gauss1("gauss1", dm, &mean1, &sigma1);
  GaussianPdf gauss2("gauss2", dm, &mean2, &sigma2);
  GaussianPdf gauss3("gauss3", dm, &mean3, &sigma3);
  ArgusPdf argus("argus", dm, &pimass, &aslope, false, &apower); 

  std::vector<Variable*> weights;
  weights.push_back(&gfrac1);
  weights.push_back(&gfrac2);
  weights.push_back(&afrac);

  std::vector<PdfBase*> comps;
  comps.push_back(&gauss1);
  comps.push_back(&gauss2);
  comps.push_back(&argus);
  comps.push_back(&gauss3);

  AddPdf resolution("resolution", weights, comps); 
  resolution.setData(data);
  FitManager mcpdf(&resolution); 

//#ifdef 0
#if 0
  #pragma omp master
  {
  std::cout << tid << "Done with data, starting minimisation" << std::endl; 
  }
#else
  std::cout << "Done with data, starting minimisation" << std::endl; 
#endif
  // Minimize
  //ROOT::Minuit2::FunctionMinimum* min = mcpdf.fit();
  mcpdf.fit(); 

  mcpdf.getMinuitValues(); 

//#ifdef 0
#if 0
#pragma omp barrier
#endif

  mean1.fixed = true;
  mean2.fixed = true;
  mean3.fixed = true;
  sigma1.fixed = true;
  sigma2.fixed = true;
  sigma3.fixed = true;
  pimass.fixed = true;
  aslope.fixed = true;
  gfrac1.fixed = true;
  gfrac2.fixed = true;
  afrac.fixed = true;
  apower.fixed = true; 

  Variable dummyzero("kpi_rd_dummyzero", 0);
  Variable delta("kpi_rd_delta", 0.000002, -0.00005, 0.00005);
  Variable epsilon("kpi_rd_epsilon", 0.05, -0.1, 0.2);
  
  ScaledGaussianPdf resolution1("resolution1", dm, &dummyzero, &sigma1, &delta, &epsilon);
  ScaledGaussianPdf resolution2("resolution2", dm, &dummyzero, &sigma2, &delta, &epsilon);
  ScaledGaussianPdf resolution3("resolution3", dm, &dummyzero, &sigma3, &delta, &epsilon);

  Variable width_bw("kpi_rd_width_bw", 0.0001, 0.00001, 0.0005);
  KinLimitBWPdf rbw1("rbw1", dm, &mean1, &width_bw);
  KinLimitBWPdf rbw2("rbw2", dm, &mean2, &width_bw);
  KinLimitBWPdf rbw3("rbw3", dm, &mean3, &width_bw);

  //#define OTHERS 1
#ifdef OTHERS
  ConvolutionPdf signal1("signal1", dm, &rbw1, &resolution1, 2); 
  ConvolutionPdf signal2("signal2", dm, &rbw2, &resolution2, 2); 
  ConvolutionPdf signal3("signal3", dm, &rbw3, &resolution3, 2); 
  std::vector<ConvolutionPdf*> convList;
  convList.push_back(&signal1); convList.push_back(&signal2); convList.push_back(&signal3);
  signal1.registerOthers(convList);
  signal2.registerOthers(convList);
  signal3.registerOthers(convList);
#else
  ConvolutionPdf signal1("signal1", dm, &rbw1, &resolution1); 
  ConvolutionPdf signal2("signal2", dm, &rbw2, &resolution2); 
  ConvolutionPdf signal3("signal3", dm, &rbw3, &resolution3); 
#endif

  signal1.setIntegrationConstants(0.1395, 0.1665, 0.0000027); 
  signal2.setIntegrationConstants(0.1395, 0.1665, 0.0000027); 
  signal3.setIntegrationConstants(0.1395, 0.1665, 0.0000027); 

 

  weights.clear();
  weights.push_back(&gfrac1);
  weights.push_back(&gfrac2);
  weights.push_back(&afrac);
 
  comps.clear();
  comps.push_back(&signal1);
  comps.push_back(&signal2);
  comps.push_back(&argus);
  comps.push_back(&signal3);
  AddPdf signal("signal", weights, comps); 

  Variable slope("kpi_rd_slope", -1.0, 0.1, -35.0, 25.0);  
  Variable *bpower = NULL;
  ArgusPdf bkg("bkg", dm, &pimass, &slope, false, bpower); 

  weights.clear();
  comps.clear();

  Variable bkg_frac("kpi_rd_bkg_frac", 0.03, 0.0, 0.3);
  weights.push_back(&bkg_frac);
  comps.push_back(&bkg);
  comps.push_back(&signal); 

//#ifdef OMP_ON
#if 0
#pragma omp master
{
#endif
  getData(); 
//#ifdef OMP_ON
#if 0
}
#pragma omp barrier
#endif

  AddPdf total("total", weights, comps);
  if (0 == fitType) total.setData(data);
  else {
    total.setData(binnedData);
    if (2 == fitType) total.setFitControl(new BinnedChisqFit()); 
  }
  FitManager datapdf(&total); 
  
//#ifdef OMP_ON
#if 0
  std::cout << tid << ": Starting fit\n"; 
#else
  std::cout << "Starting fit\n"; 
#endif
  gettimeofday(&startTime, NULL);
  startCPU = times(&startProc);
  //ROOT::Minuit2::FunctionMinimum* min2 = datapdf.fit();

<<<<<<< Updated upstream
  datapdf.fit();   
=======
  datapdf.fit();
>>>>>>> Stashed changes
  datapdf.getMinuitValues();
  std::vector<Variable*> modParams;
  total.getParameters(modParams); 
  
  std::vector<double> expected;
  expected.push_back(3.00000e-02);
  expected.push_back(1.39570e-01);
  expected.push_back(-1.00000);
  expected.push_back(5.00000e-01);
  expected.push_back(6.28037e-01);
  expected.push_back(1.90474e-02);
  expected.push_back(5.65864e-03);
  expected.push_back(1.45402e-01);
  expected.push_back(1.00000e-04);
  expected.push_back(0);
  expected.push_back(1.18496e-04);
  expected.push_back(2.00000e-06);
  expected.push_back(5.00000e-02);
  expected.push_back(1.45464e-01);
  expected.push_back(7.12482e-04);
  expected.push_back(-2.31099e+01);
  expected.push_back(1.32901);
  expected.push_back(1.45404e-01);
  expected.push_back(2.10246e-04);

  double difference;
  int count = 0;
  for (int i = 0; i < modParams.size(); i++) {
    difference = fabs(expected[i] - modParams[i]->value);
    if (difference > 0.001) {
      std::cout << "\n" << modParams[i]->name << " value not in epsilon." << endl;
      std::cout << "Expected value (compared to the GPU GooFitM value): " << expected[i] << endl;
      std::cout << "Calculated value: " << modParams[i]->value << endl;
      std::cout << "Difference: " << difference  << endl;
      count++;
    }
  }

<<<<<<< Updated upstream
  std::cout << "\nTotal differences: " << count << endl;
 
=======
  std::cout << "\nTotal differences: " << count << endl; 
>>>>>>> Stashed changes
  stopCPU = times(&stopProc);
  gettimeofday(&stopTime, NULL);

//#ifdef OMP_ON
#if 0
#pragma omp barrier
#endif
  //std::cout << "Minimum: " << *min2 << std::endl;
/*
  double dat_int = 0; 
  for (int i = 1; i <= 300; ++i) {
    dat_int += data_hist->GetBinContent(i);
  }

  signal1.setIntegrationConstants(0.1365, 0.1665, 0.00003); 
  signal2.setIntegrationConstants(0.1365, 0.1665, 0.00003); 
  signal3.setIntegrationConstants(0.1365, 0.1665, 0.00003); 
  dm->numbins = 300; 
  dm->lowerlimit = 0.1365;
  dm->upperlimit = 0.1665; 
  datapdf.getMinuitValues(); 
  std::cout << bkg_frac.value << std::endl; 

  // plotComponent seems broken? 
  TH1F* dpdf_hist = plotComponent(&total, dat_int);
  double totalIntegral = pdf_int;
  TH1F* barg_hist = plotComponent(&bkg,   dat_int*bkg_frac.value); 
  double bkgIntegral = pdf_int;
  TH1F* sign_hist = plotComponent(&signal, dat_int*(1 - bkg_frac.value));
  double sigIntegral = pdf_int; 

  double sig_int = 0; 
  double bkg_int = 0; 
  double tot_int = 0; 
  for (int i = 1; i <= 300; ++i) {
    sig_int += sign_hist->GetBinContent(i);
    bkg_int += barg_hist->GetBinContent(i); 
    tot_int += dpdf_hist->GetBinContent(i);

    dpdf_hist->SetBinContent(i, barg_hist->GetBinContent(i) + sign_hist->GetBinContent(i)); 
  } 
*/
  
//#ifdef OMP_ON
#if 0
#pragma omp master
{
#endif
  dm->value = 0.1568; 
//#ifdef OMP_ON
#if 0
}
#endif
/*
  std::cout << "PDF: " 
	    << (dat_int/totalIntegral) * total.getValue() << " " 
	    << (1-bkg_frac.value)*(dat_int/sigIntegral)*signal.getValue() << " " 
	    << bkg_frac.value*(dat_int/bkgIntegral)*bkg.getValue() << " | " 
	    << dat_int << " " << sigIntegral << " " << bkgIntegral << " " << totalIntegral << " | "
	    << sig_int << " " << bkg_int << " " << tot_int  << " | "
	    << dpdf_hist->GetBinContent(204) << " " << sign_hist->GetBinContent(204) << " " << barg_hist->GetBinContent(204) << " " 
	    << std::endl; 
*/
  
//#ifdef OMP_ON
#if 0
#pragma omp master
{
#endif
  /*
  data_hist->SetStats(false); 
  data_hist->SetMarkerStyle(8);
  data_hist->SetMarkerSize(0.6); 
  data_hist->Draw("p"); 

  dpdf_hist->SetLineColor(kViolet); 
  dpdf_hist->SetLineWidth(3); 
  dpdf_hist->Draw("lsame");
  //dpdf_hist->Draw("l");

  barg_hist->SetLineColor(kRed);
  barg_hist->SetLineWidth(3); 
  barg_hist->SetLineStyle(kDashed);
  barg_hist->Draw("lsame"); 

  sign_hist->SetLineColor(kBlue);
  sign_hist->SetLineWidth(3); 
  sign_hist->SetLineStyle(kDashed);
  sign_hist->Draw("lsame"); 

  foo->SetLogy(false); 
  foo->SaveAs("zach_linear_CUDA_fit.png"); 
  foo->SetLogy(true); 
  foo->SaveAs("zach_CUDA_fit.png"); 
  */
//#ifdef OMP_ON
#if 0
  }  // end master section
  #pragma omp barrier
}  // end parallel
#endif

}

timeval fullStart, fullStop, fullTime;

int main (int argc, char** argv) {

  gettimeofday (&fullStart, NULL);
#ifdef TARGET_MPI
  MPI_Init (&argc, &argv);
#endif

  int gpuDev = 0;   
  gStyle->SetCanvasBorderMode(0);
  gStyle->SetCanvasColor(10);
  gStyle->SetFrameFillColor(10);
  gStyle->SetFrameBorderMode(0);
  gStyle->SetPadColor(0);
  gStyle->SetTitleColor(1);
  gStyle->SetStatColor(0);
  gStyle->SetFillColor(0);
  gStyle->SetFuncWidth(1);
  gStyle->SetLineWidth(1);
  gStyle->SetLineColor(1);
  gStyle->SetPalette(1, 0);
  foo = new TCanvas(); 
 
  data_hist = new TH1F("data_hist", "", 300, 0.1365, 0.1665);
   if (argc < 2) {
     printf("Usage: zach <mode> [<device>]  \n \t mode: 0-unbinned, 1-binned, 2-binned ChiSq \n \t device is 0 by default, optionally specify GPU device other than 0\n");
     return -1;
  }
  if (argc == 3) gpuDev = atoi(argv[2]);
  CudaMinimise(gpuDev, atoi(argv[1]));  // atoi = string to integer conversion
  //RooFitMinimise(atoi(argv[3])); 

  data_hist->SetStats(false); 
  data_hist->SetMarkerStyle(8);
  data_hist->SetMarkerSize(0.6); 
  data_hist->Draw("p"); 


  // Print total minimization time
  double myCPU = stopCPU - startCPU;
  double totalCPU = myCPU; 

  timersub(&stopTime, &startTime, &totalTime);
  std::cout << "Wallclock time  : " << totalTime.tv_sec + totalTime.tv_usec/1000000.0 << " seconds." << std::endl;
  std::cout << "CPU time: " << (myCPU / CLOCKS_PER_SEC) << std::endl; 
  std::cout << "Total CPU time: " << (totalCPU / CLOCKS_PER_SEC) << std::endl; 
  myCPU = stopProc.tms_utime - startProc.tms_utime;
  std::cout << "Processor time: " << (myCPU / CLOCKS_PER_SEC) << std::endl;

  delete binnedData; 
  delete data;
  delete foo;
  delete dm;

#ifdef TARGET_MPI
  MPI_Finalize();
#endif

  gettimeofday (&fullStop, NULL);

  timersub (&fullStop, &fullStart, &fullTime);

  std::cout << "Full time: " << fullTime.tv_sec + fullTime.tv_usec/1000000.0 << " seconds." << std::endl;

  return 0; 
}
