#include "hip/hip_runtime.h"
#include "PdfBase.hh"
#include <mpi.h>
#include <typeinfo>


// This is code that belongs to the PdfBase class, that is, 
// it is common across all implementations. But it calls on device-side
// functions, and due to the nvcc translation-unit limitations, it cannot
// sit in its own object file; it must go in the CUDAglob.cu. So it's
// off on its own in this inline-cuda file, which GooPdf.cu 
// should include. 

#define CHECK_MPI(error) \
        if (error != MPI_SUCCESS) { \
					int length; \
					char message[MPI_MAX_ERROR_STRING]; \
          MPI_Error_string(error, message, &length); \
          printf("\n%.*s\n", length, message); \
          MPI_Abort(MPI_COMM_WORLD, 1);}

#ifdef CUDAPRINT
__host__ void PdfBase::copyParams (const std::vector<double>& pars) const {
  if (host_callnumber < 1) {
    std::cout << "Copying parameters: " << (long long) hipArray << " ";
  }
  for (unsigned int i = 0; i < pars.size(); ++i) {
    host_params[i] = pars[i]; 
    
    if (host_callnumber < 1) {
      std::cout << pars[i] << " ";
    }
    
    if (isnan(host_params[i])) {
      std::cout << " agh, NaN, die " << i << std::endl;
      abortWithCudaPrintFlush(__FILE__, __LINE__, "NaN in parameter"); 
    }
  }
  
  if (host_callnumber < 1) {
    std::cout << std::endl; 
  }
  MEMCPY_TO_SYMBOL(hipArray, host_params, pars.size()*sizeof(fptype), 0, hipMemcpyHostToDevice); 
}
#else 
__host__ void PdfBase::copyParams (const std::vector<double>& pars) const {
  // copyParams method performs eponymous action! 

  for (unsigned int i = 0; i < pars.size(); ++i) {
    host_params[i] = pars[i]; 
    
    if (isnan(host_params[i])) {
      std::cout << " agh, parameter is NaN, die " << i << std::endl;
      abortWithCudaPrintFlush(__FILE__, __LINE__, "NaN in parameter"); 
    }
  }

  MEMCPY_TO_SYMBOL(hipArray, host_params, pars.size()*sizeof(fptype), 0, hipMemcpyHostToDevice); 
}
#endif

__host__ void PdfBase::copyParams () {
  // Copies values of Variable objects
  parCont pars; 
  getParameters(pars); 
  std::vector<double> values; 
  for (parIter v = pars.begin(); v != pars.end(); ++v) {
    int index = (*v)->getIndex(); 
    if (index >= (int) values.size()) values.resize(index + 1);
    values[index] = (*v)->value;
  }
  copyParams(values); 
}

__host__ void PdfBase::copyNormFactors () const {
  MEMCPY_TO_SYMBOL(normalisationFactors, host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice); 
  SYNCH(); // Ensure normalisation integrals are finished
}

__host__ void PdfBase::initialiseIndices (std::vector<unsigned int> pindices) {
  // Structure of the individual index array: Number of parameters, then the indices
  // requested by the subclass (which will be interpreted by the subclass kernel), 
  // then the number of observables, then the observable indices. Notice that the
  // observable indices are not set until 'setIndices' is called, usually from setData;
  // here we only reserve space for them by setting totalParams. 
  // This is to allow index sharing between PDFs - all the PDFs must be constructed 
  // before we know what observables exist. 

  if (totalParams + pindices.size() >= maxParams) {
    std::cout << "Major problem with pindices size: " << totalParams << " + " << pindices.size() << " >= " << maxParams << std::endl; 
  }

  assert(totalParams + pindices.size() < maxParams); 
  host_indices[totalParams] = pindices.size(); 
  for (int i = 1; i <= host_indices[totalParams]; ++i) {
    host_indices[totalParams+i] = pindices[i-1]; 
  }
  host_indices[totalParams + pindices.size() + 1] = observables.size(); 
  
  parameters = totalParams;
  totalParams += (2 + pindices.size() + observables.size()); 

  /* 
  std::cout << "host_indices after " << getName() << " initialisation : ";
  for (int i = 0; i < totalParams; ++i) {
    std::cout << host_indices[i] << " ";
  }
  
  std::cout << " | " 
	    << parameters << " " 
	    << totalParams << " " 
	    << hipArray << " " 
	    << paramIndices << " "
	    << std::endl; 
  */
  MEMCPY_TO_SYMBOL(paramIndices, host_indices, totalParams*sizeof(unsigned int), 0, hipMemcpyHostToDevice); 
}

/*
__host__ void PdfBase::setData (std::vector<std::map<Variable*, fptype> >& data) {
  // Old method retained for backwards compatibility 

  if (dev_event_array) {
    gooFree(dev_event_array);
    dev_event_array = 0; 
  }

  setIndices();
  int dimensions = observables.size();
  numEntries = data.size();
  numEvents = numEntries; 
  
  fptype* host_array = new fptype[data.size()*dimensions];
  for (unsigned int i = 0; i < data.size(); ++i) {
    for (obsIter v = obsBegin(); v != obsEnd(); ++v) {
      assert(data[i].find(*v) != data[i].end()); 
      host_array[i*dimensions + (*v)->index] = data[i][*v]; 
    }
  }

  gooMalloc((void**) &dev_event_array, dimensions*numEntries*sizeof(fptype)); 
  MEMCPY(dev_event_array, host_array, dimensions*numEntries*sizeof(fptype), hipMemcpyHostToDevice);
  MEMCPY_TO_SYMBOL(functorConstants, &numEvents, sizeof(fptype), 0, hipMemcpyHostToDevice); 
  delete[] host_array; 
}
*/

__host__ void PdfBase::recursiveSetIndices () {
  for (unsigned int i = 0; i < components.size(); ++i) {
    components[i]->recursiveSetIndices(); 
  }

  int numParams = host_indices[parameters]; 
  int counter = 0; 
  for (obsIter v = obsBegin(); v != obsEnd(); ++v) {
    host_indices[parameters + 2 + numParams + counter] = (*v)->index; 
    //std::cout << getName() << " set index of " << (*v)->name << " to " << (*v)->index << " " << (parameters + 2 + numParams + counter) << std::endl; 
    counter++; 
  }  
  generateNormRange(); 
}

__host__ void PdfBase::setIndices () {
  int counter = 0; 
  for (obsIter v = obsBegin(); v != obsEnd(); ++v) {
    (*v)->index = counter++; 
  }
  recursiveSetIndices(); 
  MEMCPY_TO_SYMBOL(paramIndices, host_indices, totalParams*sizeof(unsigned int), 0, hipMemcpyHostToDevice); 

  //std::cout << "host_indices after " << getName() << " observable setIndices : ";
  //for (int i = 0; i < totalParams; ++i) {
  //std::cout << host_indices[i] << " ";
  //}
  //std::cout << std::endl; 
}

__host__ void PdfBase::setData (UnbinnedDataSet* data)
{
  //std::cout << "PdfBase::setData" << std::endl;
  if (dev_event_array) {
    gooFree(dev_event_array);
    SYNCH();
    dev_event_array = 0; 

    m_iEventsPerTask = 0;
  }

  setIndices();
  int dimensions = observables.size();
  numEntries = data->getNumEvents(); 
  numEvents = numEntries; 

#ifdef TARGET_MPI

  int world_size, world_rank;
	// Get the number of processes
  MPI_Comm_size (MPI_COMM_WORLD, &world_size);
	// Get the rank of the processes
  MPI_Comm_rank (MPI_COMM_WORLD, &world_rank); 

  int num_events_per_proc = numEntries/world_size;

  if (world_rank == 0) {
    numEvents = data->getNumEvents();
    dimensions = observables.size();
  }

  MPI_Bcast(&numEvents, sizeof(int), MPI_INT, 0, MPI_COMM_WORLD);
  MPI_Bcast(&dimensions, sizeof(int), MPI_INT, 0, MPI_COMM_WORLD);

  int *counts = new int[world_size];
  int *displacements = new int[world_size];

  //indexing for copying events over!
  for (int i = 0; i < world_size - 1; i++)
    counts[i] = num_events_per_proc;
  counts[world_size - 1] = numEntries - num_events_per_proc*(world_size - 1);
  
  //displacements into the array for indexing!
  displacements[0] = 0;
  for (int i = 1; i < world_size; i++)
    displacements[i] = displacements[i - 1] + counts[i - 1];

#endif

  fptype* host_array;

#ifdef TARGET_MPI

  if (world_rank == 0) {
    host_array = new fptype[numEntries*dimensions];
  }

  //This is an array to track if we need to redo indexing
  int fixme[observables.size ()];
  memset(fixme, 0, sizeof (int)*observables.size ());

  //printf ("Checking observables for Counts!\n");
  for (int i = 0; i < observables.size (); i++)
  {
    //printf ("%i - %s\n", i, observables[i]->name.c_str ());
    //cast this variable to see if its one we need to correct for
    CountVariable *c = dynamic_cast <CountVariable*> (observables[i]);
    //if it cast, mark it
    if (c)
    {
      fixme[i] = 1;
      //printf ("%i of %i - %s\n", i, observables.size (), c->name.c_str ());
    }
  }

	if (world_rank == 0) {
    //populate this array with our stuff
    for (int i = 0; i < numEntries; ++i)
    {
      for (obsIter v = obsBegin(); v != obsEnd(); ++v)
      {
        fptype currVal = data->getValue((*v), i);
        host_array[i*dimensions + (*v)->index] = currVal;
      }
    }

    printf("\nHost array: ");
    for (int i = 0; i < world_size; i++) {
      for (int j = 0; j < 6; j++) {	
    	  printf("%f ", host_array[displacements[i]+j]);
  	  }
    }
  }
	
#else

  host_array = new fptype[numEntries*dimensions];

  for (int i = 0; i < numEntries; ++i) 
  {
    for (obsIter v = obsBegin(); v != obsEnd(); ++v)
    {
      fptype currVal = data->getValue((*v), i);
      host_array[i*dimensions + (*v)->index] = currVal;
    }
  }

	printf("\nHost array: ");
	for (int i = 0; i < 6; i++) {
		printf("%f ", host_array[i]);
	}

#endif

#ifdef TARGET_MPI

  int mystart = displacements[world_rank];
  int myend = mystart + counts[world_rank];
  int mycount = myend - mystart;
	
	fptype *recv_buf = new fptype[mycount]; // The buffer to receive the scattered elements
	
  CHECK_MPI(MPI_Scatterv(host_array, counts, displacements, MPI_DOUBLE, recv_buf, mycount, MPI_DOUBLE, 0, MPI_COMM_WORLD));

	// Print the numbers scattered to each processor
  printf("\nProcessor rank %i out of %i processors: ", world_rank, world_size);

  for (int i = 0; i < 6; i++) {
    printf("%f ", recv_buf[i]);
  }

	// we need to fix our observables indexing to reflect having multiple cards
  for (int i = 1; i < world_size; i++)
  {
    for (int j = 0; j < counts[i]; j++)
    {
      //assumption is that the last observable is the index!
      for (int k = 0; k < dimensions; k++)
      {
        //Its counting, fix the indexing here
        if (fixme[k] > 0)
          recv_buf[j * dimensions + k] = float (j);
      }
    }
  }

#endif 

#ifdef TARGET_MPI 
	gooMalloc((void**) &dev_event_array, dimensions*mycount*sizeof(fptype));
  MEMCPY(dev_event_array, recv_buf, dimensions*mycount*sizeof(fptype), hipMemcpyHostToDevice);
  MEMCPY_TO_SYMBOL(functorConstants, &numEvents, sizeof(fptype), 0, hipMemcpyHostToDevice);
  if (world_rank == (world_size-1))  
    delete[] host_array;
  
  delete[] recv_buf; 
  printf("\ndev_event_array: ");

  for (int i = 0; i < 6; i++) {
    printf("%f ", dev_event_array[i]);
  }

  if (world_rank == (world_size - 1)) 
    printf("\n\n");

  //update everybody
  setNumPerTask(this, mycount);

  delete [] counts;
  delete [] displacements;
#else
  gooMalloc((void**) &dev_event_array, dimensions*numEntries*sizeof(fptype)); 
  MEMCPY(dev_event_array, host_array, dimensions*numEntries*sizeof(fptype), hipMemcpyHostToDevice);
  MEMCPY_TO_SYMBOL(functorConstants, &numEvents, sizeof(fptype), 0, hipMemcpyHostToDevice); 
  delete[] host_array;
#endif
}

__host__ void PdfBase::setData (BinnedDataSet* data)
{ 
  if (dev_event_array) { 
    gooFree(dev_event_array);
    dev_event_array = 0; 

    m_iEventsPerTask = 0;
  }

  setIndices();
  numEvents = 0; 
  numEntries = data->getNumBins(); 
  int dimensions = 2 + observables.size(); // Bin center (x,y, ...), bin value, and bin volume. 
  if (!fitControl->binnedFit()) setFitControl(new BinnedNllFit()); 

  fptype* host_array = new fptype[numEntries*dimensions]; 

#ifdef TARGET_MPI

	int world_size, world_rank;
  // Get the number of processes
  MPI_Comm_size (MPI_COMM_WORLD, &world_size);
  // Get the rank of the processes
  MPI_Comm_rank (MPI_COMM_WORLD, &world_rank);

  //This is an array to track if we need to redo indexing
  int fixme[dimensions];
  memset(fixme, 0, sizeof (int)*dimensions);

  for (int i = 0; i < observables.size (); i++)
  {
    //cast this variable to see if its one we need to correct for
    CountVariable *c = dynamic_cast <CountVariable*> (observables[i]);
    //if it cast, mark it
    if (c)
      fixme[i] = 1;
  }

	if (world_rank == 0) { 
    // populate the array
    for (unsigned int i = 0; i < numEntries; ++i) {
      for (obsIter v = obsBegin(); v != obsEnd(); ++v) {
        host_array[i*dimensions + (*v)->index] = data->getBinCenter((*v), i);
      }

      host_array[i*dimensions + observables.size() + 0] = data->getBinContent(i);
      host_array[i*dimensions + observables.size() + 1] = fitControl->binErrors() ? data->getBinError(i) : data->getBinVolume(i);
      numEvents += data->getBinContent(i);
    }

		printf("\nHost array: ");
  	for (int i = 0; i < 25; i++) {
    	printf("%f ", host_array[i]);
  	}
  }

#else

	// populate the array
  for (unsigned int i = 0; i < numEntries; ++i) {
  	  for (obsIter v = obsBegin(); v != obsEnd(); ++v) {
    	  host_array[i*dimensions + (*v)->index] = data->getBinCenter((*v), i); 
    	}
		
			host_array[i*dimensions + observables.size() + 0] = data->getBinContent(i);
			host_array[i*dimensions + observables.size() + 1] = fitControl->binErrors() ? data->getBinError(i) : data->getBinVolume(i); 
			numEvents += data->getBinContent(i);	
	}

	printf("\nHost array: ");
  for (int i = 0; i < 25; i++) {
    printf("%f ", host_array[i]);
  }

#endif

#if TARGET_MPI

  int num_events_per_proc = numEvents/world_size;

  int *counts = new int[world_size];
  int *displacements = new int[world_size];

  //indexing for copying events over!
  for (int i = 0; i < world_size - 1; i++)
    counts[i] = num_events_per_proc;
  counts[world_size - 1] = numEvents - num_events_per_proc*(world_size - 1);

  //displacements into the array for indexing!
  displacements[0] = 0;
  for (int i = 1; i < world_size; i++)
    displacements[i] = displacements[i - 1] + counts[i - 1];

  int mystart = displacements[world_rank];
  int myend = mystart + counts[world_rank];
  int mycount = myend - mystart;
	
	fptype *recv_buf = new fptype[mycount];

	// check data type of fptype
	if (typeid(fptype) == typeid(float)) {
		printf("\nfloat");
	  CHECK_MPI(MPI_Scatterv(host_array, counts, displacements, MPI_FLOAT, recv_buf, mycount, MPI_FLOAT, 0, MPI_COMM_WORLD));
	} else {
		// fptype is of type double
		printf("\ndouble");
		CHECK_MPI(MPI_Scatterv(host_array, counts, displacements, MPI_DOUBLE, recv_buf, mycount, MPI_DOUBLE, 0, MPI_COMM_WORLD));
	}

  // Print the numbers scattered to each processor
  printf("\nProcessor rank %d"
           " out of %d processors: ", world_rank, world_size);

	// we need to fix our observables indexing to reflect having multiple cards
  for (int i = 1; i < world_size; i++)
  {
    for (int j = 0; j < counts[i]; j++)
    {
      //assumption is that the last observable is the index!
      for (int k = 0; k < dimensions; k++)
      {
        //Its counting, fix the indexing here
        if (fixme[k] > 0)
          recv_buf[(j + displacements[i])*dimensions + dimensions - k] = float (j);
      }
    }
  }

	for (int i = 0; i < 25; i++) {
    printf("%f ", recv_buf[i]);
  }

#endif

#ifdef TARGET_MPI
  gooMalloc((void**) &dev_event_array, dimensions*mycount*sizeof(fptype)); 
  MEMCPY(dev_event_array, recv_buf, dimensions*mycount*sizeof(fptype), hipMemcpyHostToDevice); 
  MEMCPY_TO_SYMBOL(functorConstants, &numEvents, sizeof(fptype), 0, hipMemcpyHostToDevice); 
  delete[] host_array;
	delete[] recv_buf;

  //update our displacements:
  for (int i = 0; i < world_size; i++)
    displacements[i] = 0;

  //update everybody
  setNumPerTask(this, mycount);

  delete [] counts;
  delete [] displacements;
#else
  gooMalloc((void**) &dev_event_array, dimensions*numEntries*sizeof(fptype)); 
  MEMCPY(dev_event_array, host_array, dimensions*numEntries*sizeof(fptype), hipMemcpyHostToDevice);
#endif
}

__host__ void PdfBase::generateNormRange () {
  if (normRanges) gooFree(normRanges);
  gooMalloc((void**) &normRanges, 3*observables.size()*sizeof(fptype));
  
  fptype* host_norms = new fptype[3*observables.size()];
  int counter = 0; // Don't use index in this case to allow for, eg, 
  // a single observable whose index is 1; or two observables with indices
  // 0 and 2. Make one array per functor, as opposed to variable, to make
  // it easy to pass MetricTaker a range without worrying about which parts
  // to use. 
  for (obsIter v = obsBegin(); v != obsEnd(); ++v) {
    host_norms[3*counter+0] = (*v)->lowerlimit;
    host_norms[3*counter+1] = (*v)->upperlimit;
    host_norms[3*counter+2] = integrationBins > 0 ? integrationBins : (*v)->numbins;
    counter++; 
  }

  MEMCPY(normRanges, host_norms, 3*observables.size()*sizeof(fptype), hipMemcpyHostToDevice);
  delete[] host_norms; 
}

void PdfBase::clearCurrentFit () {
  totalParams = 0; 
  gooFree(dev_event_array);
  dev_event_array = 0; 
}

__host__ void PdfBase::printProfileInfo (bool topLevel) {
#ifdef PROFILING
  if (topLevel) {
    hipError_t err = MEMCPY_FROM_SYMBOL(host_timeHist, timeHistogram, 10000*sizeof(fptype), 0);
    if (hipSuccess != err) {
      std::cout << "Error on copying timeHistogram: " << hipGetErrorString(err) << std::endl;
      return;
    }
    
    std::cout << getName() << " : " << getFunctionIndex() << " " << host_timeHist[100*getFunctionIndex() + getParameterIndex()] << std::endl; 
    for (unsigned int i = 0; i < components.size(); ++i) {
      components[i]->printProfileInfo(false); 
    }
  }
#endif
}



gooError gooMalloc (void** target, size_t bytes) {
// Thrust 1.7 will make the use of THRUST_DEVICE_BACKEND an error
#if THRUST_DEVICE_BACKEND==THRUST_DEVICE_BACKEND_OMP || THRUST_DEVICE_SYSTEM==THRUST_DEVICE_BACKEND_OMP
  target[0] = malloc(bytes);
  if (target[0]) return gooSuccess;
  else return gooErrorMemoryAllocation; 
#else
  return (gooError) hipMalloc(target, bytes); 
#endif
}

gooError gooFree (void* ptr) {
// Thrust 1.7 will make the use of THRUST_DEVICE_BACKEND an error
#if THRUST_DEVICE_BACKEND==THRUST_DEVICE_BACKEND_OMP || THRUST_DEVICE_SYSTEM==THRUST_DEVICE_BACKEND_OMP
  free(ptr);
  return gooSuccess;
#else
  return (gooError) hipFree(ptr); 
#endif
}
